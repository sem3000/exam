#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;


__global__ void add(int *a)
{
int tid = threadIdx.x;
int no_of_threads = blockDim.x;
int step=1;

while(no_of_threads>0)
{
if(tid<no_of_threads)
{
	int first = tid*step*2;
	int second = first+step;
	
	a[first]+=a[second];
}
step<<=1;
no_of_threads>>=1;
}

}

__global__ void max(int *a)
{
int tid=threadIdx.x;
int step =1;
int no_of_threads = blockDim.x;

while(no_of_threads>0)
{
if(tid<no_of_threads)
	{
	int first = tid*step*2;
	int second = first+step;
	
	a[first] = a[first]>a[second]?a[first]:a[second];
	}
	step<<=1;
	no_of_threads>>=1;
}
}


__global__ void min(int *a)
{
int tid=threadIdx.x;
int step =1;
int no_of_threads = blockDim.x;

while(no_of_threads>0)
{
if(tid<no_of_threads)
	{
	int first = tid*step*2;
	int second = first+step;
	
	a[first] = a[first]<a[second]?a[first]:a[second];
	}
	step<<=1;
	no_of_threads>>=1;
}
}

__global__ void stdDev(int *a,int mean){
a[threadIdx.x]-=mean;
a[threadIdx.x]*=a[threadIdx.x];
}
        
int main()
{
int host_arr[]={1,2,3,4,5,6,7,8};
int *dev_arr;
int SIZE=8;

hipMalloc((void**)&dev_arr,SIZE*sizeof(int));

//SUM AND AVERAGE
hipMemcpy(dev_arr,host_arr,SIZE*sizeof(int),hipMemcpyHostToDevice);
add<<<1,SIZE/2>>>(dev_arr);

int sum;
hipMemcpy(&sum,dev_arr,sizeof(int),hipMemcpyDeviceToHost);

int mean=sum/SIZE;

cout<<"Sum is : "<<sum;
cout<<"Average is : "<<mean;

//MAX
hipMemcpy(dev_arr,host_arr,SIZE*sizeof(int),hipMemcpyHostToDevice);
max<<<1,SIZE/2>>>(dev_arr);

int max;
hipMemcpy(&max,dev_arr,sizeof(int),hipMemcpyDeviceToHost);

cout<<"Max is : "<<max;

//MIN

hipMemcpy(dev_arr,host_arr,SIZE*sizeof(int),hipMemcpyHostToDevice);
min<<<1,SIZE/2>>>(dev_arr);

int min;
hipMemcpy(&min,dev_arr,sizeof(int),hipMemcpyDeviceToHost);

cout<<"Min is : "<<min;

cout<<"\n\n";
//STDDV
hipMemcpy(dev_arr,host_arr,SIZE*sizeof(int),hipMemcpyHostToDevice);
stdDev<<<1,SIZE>>>(dev_arr,mean);
hipMemcpy(host_arr,dev_arr,SIZE*sizeof(int),hipMemcpyDeviceToHost);
cout<<host_arr[0];
cout<<host_arr[1];
cout<<host_arr[2];
cout<<host_arr[3];
cout<<host_arr[4];
cout<<host_arr[5];
cout<<host_arr[6];
cout<<host_arr[7];

cout<<"\n\n";

hipMemcpy(dev_arr,host_arr,SIZE*sizeof(int),hipMemcpyHostToDevice);
add<<<1,SIZE/2>>>(dev_arr);
int stdDeviation;
hipMemcpy(&stdDeviation,dev_arr,sizeof(int),hipMemcpyDeviceToHost);
cout<<"STDDEV:"<<sqrt(stdDeviation/SIZE); 

}
