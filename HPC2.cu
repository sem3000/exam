#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define SIZE 100
#define n 2

using namespace std;

__global__ void vec_add(int *x,int *y,int *z)
{
int id = blockIdx.x*blockDim.x+threadIdx.x;
z[id]=x[id]+y[id];
}

__global__ void vec_mat_mul(int *mat,int *vec,int *o)
{
int x = threadIdx.x;
printf("\n%d",x);


o[x]=0;
for(int k=0;k<n;k++)
o[x]=o[x]+vec[k]*mat[k*n+k];

}

__global__ void mat_mul(int *a,int *b,int* c)
{
int x = threadIdx.x;
int y = threadIdx.y;

c[n*y+x]=0; //here col2
  for(int k=0;k<n;k++) //here col1
    c[n*y+x]=c[n*y+x]+a[n*y+k]*b[n*k+x];  //col2,col2,col1,col2
}


int main()
{
//vec_add
int a[SIZE],b[SIZE],c[SIZE];
int *d,*e,*f;

for(int i=0;i<SIZE;i++)
a[i]=b[i]=i;

hipMalloc((void**)&d,SIZE*sizeof(int));
hipMalloc((void**)&e,SIZE*sizeof(int));
hipMalloc((void**)&f,SIZE*sizeof(int));

hipMemcpy(d,a,SIZE*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(e,b,SIZE*sizeof(int),hipMemcpyHostToDevice);

vec_add<<<5,20>>>(d,e,f);

hipMemcpy(c,f,SIZE*sizeof(int),hipMemcpyDeviceToHost);

printf("%d",c[50]);

//mat_mul
int mat1[n][n],mat2[n][n],mat3[n][n];
int *g,*h,*l;

for(int i=0;i<n;i++){
for(int j=0;j<n;j++){
mat1[i][j]=mat2[i][j]=1;
}}

hipMalloc((void**)&g,n*n*sizeof(int));
hipMalloc((void**)&h,n*n*sizeof(int));
hipMalloc((void**)&l,n*n*sizeof(int));

hipMemcpy(g,mat1,n*n*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(h,mat2,n*n*sizeof(int),hipMemcpyHostToDevice);

dim3 threadBlock(n,n); //col2 row1
mat_mul<<<1,threadBlock>>>(g,h,l);

hipMemcpy(mat3,l,n*n*sizeof(int),hipMemcpyDeviceToHost);

for(int i=0;i<n;i++){
for(int j=0;j<n;j++){
printf("%d",mat3[i][j]);
}}


int mat4[n][n];
int *w;
int vec4[n],o4[n];
int *u,*out4;

for(int i=0;i<n;i++){
for(int j=0;j<n;j++){
mat4[i][j]=1;
}}

for(int i=0;i<n;i++)
vec4[i]=1;

hipMalloc((void**)&w,n*n*sizeof(int));
hipMalloc((void**)&u,n*sizeof(int));
hipMalloc((void**)&out4,n*sizeof(int));

hipMemcpy(w,mat4,n*n*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(u,vec4,n*sizeof(int),hipMemcpyHostToDevice);

vec_mat_mul<<<1,n>>>(w,u,out4);

hipMemcpy(o4,out4,n*sizeof(int),hipMemcpyDeviceToHost);

cout<<"\n\n";
for(int i=0;i<n;i++)
cout<<o4[i];


return 0;
}
